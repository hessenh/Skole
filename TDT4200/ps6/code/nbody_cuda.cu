#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>

#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 32

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays 
float2* velocities_d;
float4* planets_d;

// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }
    
    num_timesteps = strtol(argv[1], 0, 10);
}

double walltime ( void ) {
    static struct timeval t;
    gettimeofday ( &t, NULL );
    return ( t.tv_sec + 1e-6 * t.tv_usec );
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){
    float2 dv;
    float2 dist;
    dist.x = q.x - p.x;
    dist.y = q.y - p.y;


    float abs_dist = sqrt(dist.x*dist.x + dist.y*dist.y);
    float dist_cubed = abs_dist*abs_dist*abs_dist;
    //printf("%f %f\n", abs_dist, dist_cubed);



    dv.x = dT*G*q.z/dist_cubed * dist.x;
    dv.y = dT*G*q.z/dist_cubed * dist.y;
    return dv;
}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){
    float2 velocityChange;
    velocityChange.x = 0;
    velocityChange.y = 0;

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        if (my_planet.x == shared_planets[i].x && my_planet.y == shared_planets[i].y) continue;
        float2 newChange = calculate_velocity_change_planet(my_planet, shared_planets[i]);
        velocityChange.x += newChange.x;
        velocityChange.y += newChange.y;
    }

    return velocityChange;
}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    __shared__ float4 shared[BLOCK_SIZE];
    float4 planet = planets[tid];
    float2 velocityChange;
    velocityChange.x = velocities[tid].x;
    velocityChange.y = velocities[tid].y;

    for (int i = 0; i < num_planets; i+=BLOCK_SIZE)
    {
        shared[threadIdx.x] = planets[threadIdx.x + i];
        __syncthreads();
        float2 vc = calculate_velocity_change_block(planet, shared);
        velocityChange.x += vc.x;
        velocityChange.y += vc.y;
        __syncthreads();
    }
    velocities[tid].x = velocityChange.x;
    velocities[tid].y = velocityChange.y;
}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    planets[tid].x += velocities[tid].x * dT;
    planets[tid].y += velocities[tid].y * dT;
}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device 
    int error;

    double start=walltime();
    
    /* Allocate device memory, and point to it from local variable */
    error = hipMalloc((void**)&planets_d, sizeof(float4)*num_planets);
    /* Everything OK? */
    if (error != hipSuccess)
        printf("Malloc: %d\n", error);

    error = hipMalloc((void**)&velocities_d, sizeof(float2)*num_planets);
    /* Everything OK? */
    if (error != hipSuccess)
        printf("Malloc: %d\n", error);
    
    double mallocTime=walltime();

    /* We transfer memory like this */
    error = hipMemcpy(planets_d, planets, sizeof(float4)*num_planets, hipMemcpyHostToDevice);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy planets to device: %d\n", error);

    error = hipMemcpy(velocities_d, velocities, sizeof(float2)*num_planets, hipMemcpyHostToDevice);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy to velocities to device: %d\n", error);

    double memTime = walltime();

    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
        update_velocities<<<num_blocks,BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
        // We print the error code from last call if things went bad
        error = hipGetLastError();
        if (error != hipSuccess)
            printf("update_velocities error - Step: %d CUDA Code: %d\n", t, error);
        hipDeviceSynchronize();
        update_positions<<<num_blocks,BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
        // We print the error code from last call if things went bad
        error = hipGetLastError();
        if (error != hipSuccess)
            printf("update_positions errror - Step: %d CUDA Code: %d\n", t, error);
        hipDeviceSynchronize();

    }
    double calcTime = walltime();

    // TODO 3. Transfer data back to host
    error = hipMemcpy(planets, planets_d, sizeof(float4)*num_planets, hipMemcpyDeviceToHost);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy to planets back to host: %d\n", error);

    error = hipMemcpy(velocities, velocities_d, sizeof(float2)*num_planets, hipMemcpyDeviceToHost);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy velocities back to host: %d\n", error);

    hipFree(planets_d);
    hipFree(velocities_d);

    double tranferBackTime = walltime();

    hipDeviceSynchronize();
    printf("Malloc device time: %f\n", mallocTime - start);
    printf("Copy to device time: %f\n", memTime - mallocTime);
    printf("Calc time: %f\n", calcTime - memTime);
    printf("Copy to host time: %f\n", tranferBackTime - calcTime);
    printf("Total time: %f\n", walltime() - start);
    // Output
    write_planets(num_timesteps);
}
