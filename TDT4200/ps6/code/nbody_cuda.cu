#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 64

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays 
float2* velocities_d;
float4* planets_d;


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }
    
    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){

}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){

}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){

}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){

}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device 
    int error;

    /* Allocate device memory, and point to it from local variable */
    error = hipMalloc((void**)&planets_d, sizeof(float4)*num_planets);
    /* Everything OK? */
    if (error != hipSuccess)
        printf("Malloc: %d\n", error);

    error = hipMalloc((void**)&velocities_d, sizeof(float2)*num_planets);
    /* Everything OK? */
    if (error != hipSuccess)
        printf("Malloc: %d\n", error);

    /* We transfer memory like this */
    error = hipMemcpy(planets_d, planets, sizeof(float4)*num_planets, hipMemcpyHostToDevice);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy planets to device: %d\n", error);

    error = hipMemcpy(velocities_d, velocities, sizeof(float2)*num_planets, hipMemcpyHostToDevice);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy to velocities to device: %d\n", error);
  
    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
        update_velocities<<<num_blocks,BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
        // We print the error code from last call if things went bad
        error = hipGetLastError();
        if (error != hipSuccess)
            printf("update_velocities error - Step: %d CUDA Code: %d\n", t, error);

        update_positions<<<num_blocks,BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
        // We print the error code from last call if things went bad
        error = hipGetLastError();
        if (error != hipSuccess)
            printf("update_positions errror - Step: %d CUDA Code: %d\n", t, error);

    }

    // TODO 3. Transfer data back to host
    error = hipMemcpy(planets, planets_d, sizeof(float4)*num_planets, hipMemcpyDeviceToHost);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy to planets back to host: %d\n", error);

    error = hipMemcpy(velocities, velocities_d, sizeof(float2)*num_planets, hipMemcpyDeviceToHost);
    /* and check for errors */
    if (error != hipSuccess)
        printf("Copy velocities back to host: %d\n", error);

    hipFree(planets_d);
    hipFree(velocities_d);

    // Output
    write_planets(num_timesteps);
}
