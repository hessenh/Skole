#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

/* Problem size */
#define XSIZE 2560
#define YSIZE 2048

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 128
#define BLOCKX 160

#define MAXITER 255

double xleft=-2.01;
double xright=1;
double yupper,ylower;
double ycenter=1e-6;
double step;

int host_pixel[XSIZE*YSIZE];
int device_pixel[XSIZE*YSIZE];

typedef struct {
	double real,imag;
} my_complex_t;

#define PIXEL(i,j) ((i)+(j)*XSIZE)

/********** SUBTASK1: Create kernel device_calculate *************************/

/*
 * CUDA kernel for calculation the same as host_calculate, the Mandelbrot-set
 */
__global__ void device_calculate(int *pixel, float xleft, float yupper, float step) {
    // In the CUDA-edition, we find i and j from the thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    /* Calculate the number of iterations until divergence for each pixel.
       If divergence never happens, return MAXITER */
    
    // Didn't get the struct to work properly, so I split it
    float cr,ci,zr,zi,tempr,tempi;
    int iter=0;
    cr = (xleft + step*i);
    ci = (yupper - step*j);
    zr = cr;
    zi = ci;
    while(zr*zr + zi*zi < 4.0) {
        tempr = zr*zr - zi*zi + cr;
        tempi = 2.0*zr*zi + ci;
        zr = tempr;
        zi = tempi;
        if(++iter==MAXITER) break;
    }
    pixel[PIXEL(i,j)]=iter;

}

/********** SUBTASK1 END *****************************************************/

void host_calculate() {
    for(int j=0;j<YSIZE;j++) {
        for(int i=0;i<XSIZE;i++) {
            /* Calculate the number of iterations until divergence for each pixel.
               If divergence never happens, return MAXITER */
            float cr,ci,zr,zi,tempr,tempi;
            int iter=0;
            cr = (xleft + step*i);
            ci = (yupper - step*j);
            zr = cr;
            zi = ci;
            while(zr*zr + zi*zi < 4.0) {
                tempr = zr*zr - zi*zi + cr;
                tempi = 2.0*zr*zi + ci;
                zr = tempr;
                zi = tempi;
                if(++iter==MAXITER) break;
            }
            host_pixel[PIXEL(i,j)]=iter;
        }
    }
}

typedef unsigned char uchar;

/* save 24-bits bmp file, buffer must be in bmp format: upside-down */
void savebmp(char *name,uchar *buffer,int x,int y) {
    FILE *f=fopen(name,"wb");
    if(!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size=x*y*3+54;
    uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,
        0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,
        0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    fwrite(header,1,54,f);
    fwrite(buffer,1,x*y*3,f);
    fclose(f);
}

/* given iteration number, set a colour */
void fancycolour(uchar *p,int iter) {
    if(iter==MAXITER);
    else if(iter<8) { p[0]=128+iter*16; p[1]=p[2]=0; }
    else if(iter<24) { p[0]=255; p[1]=p[2]=(iter-8)*16; }
    else if(iter<160) { p[0]=p[1]=255-(iter-24)*2; p[2]=255; }
    else { p[0]=p[1]=(iter-160)*2; p[2]=255-(iter-160)*2; }
}

/*
 * Get system time to microsecond precision (ostensibly, the same as MPI_Wtime),
 * returns time in seconds
 */
double walltime ( void ) {
    static struct timeval t;
    gettimeofday ( &t, NULL );
    return ( t.tv_sec + 1e-6 * t.tv_usec );
}

int main(int argc,char **argv) {
    if(argc==1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1=yes, 0=no)");
        return 0;
    }
    double start;
    double hosttime=0;
    double devicetime=0;
    double memtime=0;

    hipDeviceProp_t p;
    hipSetDevice(0);
    hipGetDeviceProperties (&p, 0);
    printf("Device compute capability: %d.%d\n", p.major, p.minor);

    /* Calculate the range in the y-axis such that we preserve the
       aspect ratio */
    step=(xright-xleft)/XSIZE;
    yupper=ycenter+(step*YSIZE)/2;
    ylower=ycenter-(step*YSIZE)/2;

    /* Host calculates image */
    start=walltime();
    host_calculate();
    hosttime+=walltime()-start;

    /********** SUBTASK2: Set up device memory *******************************/

    // Put device pixel memory pointer on local stack
    int *dev_pixel;
    // Place for error codes
    int error;

    // Allocate device memory, and point to it from local variable
    error = hipMalloc((void**)&dev_pixel, sizeof(int)*XSIZE*YSIZE);

    // Everything OK?
    if (error != hipSuccess)
        printf("Malloc: %d\n", error);

    /********** SUBTASK2 END *************************************************/

    start=walltime();
    /********** SUBTASK3: Execute the kernel on the device *******************/

    // Find out how many threads and blocks we need by
    // looking at block size and space size
    dim3 threads;
    dim3 blocks;
    blocks.x = BLOCKX;
    blocks.y = BLOCKY;
    threads.x = XSIZE / BLOCKX;
    threads.y = YSIZE / BLOCKY;

    device_calculate<<<blocks,threads>>>(dev_pixel, xleft, yupper, step);
    // We print the error code from last call if things went bad
    error = hipGetLastError();
    if (error != hipSuccess)
        printf("CUDA Code: %d\n", error);

    // Since kernel calling is asyncronus, and we time the calculation, it would be unfare not to syncronize here
    hipDeviceSynchronize();

    /********** SUBTASK3 END *************************************************/

    devicetime+=walltime()-start;

    start=walltime();

    /********** SUBTASK4: Transfer the result from device to device_pixel[][]*/

    // We transfer memory like this
    error = hipMemcpy(device_pixel, dev_pixel, sizeof(int)*XSIZE*YSIZE, hipMemcpyDeviceToHost);
    // and check for errors
    if (error != hipSuccess)
        printf("Copy to host: %d\n", error);

    /********** SUBTASK4 END *************************************************/

    memtime+=walltime()-start;

    /********** SUBTASK5: Free the device memory also ************************/

    // Finally we free the device memory
    error = hipFree(dev_pixel);
    // and check for errors
    if (error != hipSuccess)
        printf("Free device memory: %d\n", error);

    /********** SUBTASK5 END *************************************************/

    int errors=0;
    /* check if result is correct */
    for(int i=0;i<XSIZE;i++) {
        for(int j=0;j<YSIZE;j++) {
            int diff=host_pixel[PIXEL(i,j)]-device_pixel[PIXEL(i,j)];
            if(diff<0) diff=-diff;
            /* allow +-1 difference */
            if(diff>1) {
                if(errors<10) printf("Error on pixel %d %d: expected %d, found %d\n",
                        i,j,host_pixel[PIXEL(i,j)],device_pixel[PIXEL(i,j)]);
                else if(errors==10) puts("...");
                errors++;
            }
        }
    }
    if(errors>0) printf("Found %d errors.\n",errors);
    else puts("Device calculations are correct.");

    printf("\n");
    printf("Host time:          %7.3f ms\n",hosttime*1e3);
    printf("Device calculation: %7.3f ms\n",devicetime*1e3);
    printf("Copy result:        %7.3f ms\n",memtime*1e3);

    if(strtol(argv[1],NULL,10)!=0) {
        /* create nice image from iteration counts. take care to create it upside
           down (bmp format) */
        unsigned char *buffer=(unsigned char *)calloc(XSIZE*YSIZE*3,1);
        for(int i=0;i<XSIZE;i++) {
            for(int j=0;j<YSIZE;j++) {
                int p=((YSIZE-j-1)*XSIZE+i)*3;
                fancycolour(buffer+p,device_pixel[PIXEL(i,j)]);
            }
        }
        /* write image to disk */
        savebmp("mandel1.bmp",buffer,XSIZE,YSIZE);
    }
    return 0;
}
